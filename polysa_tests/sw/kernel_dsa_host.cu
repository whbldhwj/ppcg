#include <assert.h>
#include <stdio.h>
#include "kernel_dsa_kernel.hu"
#include "kernel.h"

void dsa_kernel(char alt[M], char ref[N], int H[M + 1][N + 1], int bt[M][N]) {
  char alt_ext[M][N];
  char ref_ext[M][N];
  int sim_score_ext[M][N];
  int H_ext[M][N];

  int step_diag_ext[M][N];
 
  int best_gap_v_ext[M][N];
  int gap_size_v_ext[M][N];
  int step_down_ext[M][N];
  int kd_ext[M][N];
  
  int best_gap_h_ext[M][N];
  int gap_size_h_ext[M][N];
  int step_right_ext[M][N];
  int ki_ext[M][N];

  int sw_tmp1_ext[M][N];
  int sw_tmp2_ext[M][N];
  int bt_tmp1_ext[M][N];
  int bt_tmp2_ext[M][N];

  {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

    int *dev_H;
    int *dev_H_ext;
    char *dev_alt;
    char *dev_alt_ext;
    int *dev_best_gap_h_ext;
    int *dev_best_gap_v_ext;
    int *dev_bt;
    int *dev_gap_size_h_ext;
    int *dev_gap_size_v_ext;
    char *dev_ref;
    char *dev_ref_ext;
    int *dev_sim_score_ext;
    int *dev_step_diag_ext;
    
    cudaCheckReturn(hipMalloc((void **) &dev_H, (17) * (9) * sizeof(int)));
    cudaCheckReturn(hipMalloc((void **) &dev_H_ext, (16) * (8) * sizeof(int)));
    cudaCheckReturn(hipMalloc((void **) &dev_alt, (16) * sizeof(char)));
    cudaCheckReturn(hipMalloc((void **) &dev_alt_ext, (16) * (8) * sizeof(char)));
    cudaCheckReturn(hipMalloc((void **) &dev_best_gap_h_ext, (16) * (8) * sizeof(int)));
    cudaCheckReturn(hipMalloc((void **) &dev_best_gap_v_ext, (16) * (8) * sizeof(int)));
    cudaCheckReturn(hipMalloc((void **) &dev_bt, (16) * (8) * sizeof(int)));
    cudaCheckReturn(hipMalloc((void **) &dev_gap_size_h_ext, (16) * (8) * sizeof(int)));
    cudaCheckReturn(hipMalloc((void **) &dev_gap_size_v_ext, (16) * (8) * sizeof(int)));
    cudaCheckReturn(hipMalloc((void **) &dev_ref, (8) * sizeof(char)));
    cudaCheckReturn(hipMalloc((void **) &dev_ref_ext, (16) * (8) * sizeof(char)));
    cudaCheckReturn(hipMalloc((void **) &dev_sim_score_ext, (16) * (8) * sizeof(int)));
    cudaCheckReturn(hipMalloc((void **) &dev_step_diag_ext, (16) * (8) * sizeof(int)));
    
    cudaCheckReturn(hipMemcpy(dev_H, H, (17) * (9) * sizeof(int), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_alt, alt, (16) * sizeof(char), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_best_gap_h_ext, best_gap_h_ext, (16) * (8) * sizeof(int), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_best_gap_v_ext, best_gap_v_ext, (16) * (8) * sizeof(int), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_gap_size_h_ext, gap_size_h_ext, (16) * (8) * sizeof(int), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_gap_size_v_ext, gap_size_v_ext, (16) * (8) * sizeof(int), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_ref, ref, (8) * sizeof(char), hipMemcpyHostToDevice));
    {
      dim3 k0_dimBlock(8);
      dim3 k0_dimGrid(1);
      kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_best_gap_v_ext);
      cudaCheckKernel();
    }
    
    {
      dim3 k1_dimBlock(16);
      dim3 k1_dimGrid(1);
      kernel1 <<<k1_dimGrid, k1_dimBlock>>> (dev_alt, dev_alt_ext);
      cudaCheckKernel();
    }
    
    {
      dim3 k2_dimBlock(8);
      dim3 k2_dimGrid(1);
      kernel2 <<<k2_dimGrid, k2_dimBlock>>> (dev_gap_size_v_ext);
      cudaCheckKernel();
    }
    
    {
      dim3 k3_dimBlock(8);
      dim3 k3_dimGrid(1);
      kernel3 <<<k3_dimGrid, k3_dimBlock>>> (dev_ref, dev_ref_ext);
      cudaCheckKernel();
    }
    
    {
      dim3 k4_dimBlock(16);
      dim3 k4_dimGrid(1);
      kernel4 <<<k4_dimGrid, k4_dimBlock>>> (dev_alt_ext);
      cudaCheckKernel();
    }
    
    {
      dim3 k5_dimBlock(8);
      dim3 k5_dimGrid(1);
      kernel5 <<<k5_dimGrid, k5_dimBlock>>> (dev_ref_ext);
      cudaCheckKernel();
    }
    
    {
      dim3 k6_dimBlock(8, 16);
      dim3 k6_dimGrid(1, 1);
      kernel6 <<<k6_dimGrid, k6_dimBlock>>> (dev_alt_ext, dev_ref_ext, dev_sim_score_ext, dev_step_diag_ext);
      cudaCheckKernel();
    }
    
    {
      dim3 k7_dimBlock(16);
      dim3 k7_dimGrid(1);
      kernel7 <<<k7_dimGrid, k7_dimBlock>>> (dev_best_gap_h_ext);
      cudaCheckKernel();
    }
    
    {
      dim3 k8_dimBlock(16);
      dim3 k8_dimGrid(1);
      kernel8 <<<k8_dimGrid, k8_dimBlock>>> (dev_gap_size_h_ext);
      cudaCheckKernel();
    }
    
    for (int c0 = 0; c0 <= 22; c0 += 1)
      {
        dim3 k9_dimBlock(16);
        dim3 k9_dimGrid(1);
        kernel9 <<<k9_dimGrid, k9_dimBlock>>> (dev_H, dev_H_ext, dev_best_gap_h_ext, dev_best_gap_v_ext, dev_bt, dev_gap_size_h_ext, dev_gap_size_v_ext, dev_sim_score_ext, dev_step_diag_ext, c0);
        cudaCheckKernel();
      }
      
    cudaCheckReturn(hipMemcpy(H, dev_H, (17) * (9) * sizeof(int), hipMemcpyDeviceToHost));
    cudaCheckReturn(hipMemcpy(bt, dev_bt, (16) * (8) * sizeof(int), hipMemcpyDeviceToHost));
    cudaCheckReturn(hipFree(dev_H));
    cudaCheckReturn(hipFree(dev_H_ext));
    cudaCheckReturn(hipFree(dev_alt));
    cudaCheckReturn(hipFree(dev_alt_ext));
    cudaCheckReturn(hipFree(dev_best_gap_h_ext));
    cudaCheckReturn(hipFree(dev_best_gap_v_ext));
    cudaCheckReturn(hipFree(dev_bt));
    cudaCheckReturn(hipFree(dev_gap_size_h_ext));
    cudaCheckReturn(hipFree(dev_gap_size_v_ext));
    cudaCheckReturn(hipFree(dev_ref));
    cudaCheckReturn(hipFree(dev_ref_ext));
    cudaCheckReturn(hipFree(dev_sim_score_ext));
    cudaCheckReturn(hipFree(dev_step_diag_ext));
  }
}
