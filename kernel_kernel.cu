#include "hip/hip_runtime.h"
#include "kernel_kernel.hu"
__global__ void kernel0(char *alt, char *ref, int *sim_score_ext, int *step_diag_ext)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ char shared_alt[16];
    __shared__ char shared_ref[8];
    int private_sim_score_ext[1][1];

    {
      if (t0 == 0) {
        for (int c0 = t1; c0 <= 15; c0 += 8)
          shared_alt[c0] = alt[c0];
        shared_ref[t1] = ref[t1];
      }
      __syncthreads();
      private_sim_score_ext[0][0] = ((shared_alt[t0] == shared_ref[t1]) ? 200 : (-150));
      if (t0 == 0) {
        step_diag_ext[0 * 8 + t1] = (((-100000000) > private_sim_score_ext[0][0]) ? (-100000000) : private_sim_score_ext[0][0]);
      } else if (t1 == 0) {
        step_diag_ext[t0 * 8 + 0] = (((-100000000) > private_sim_score_ext[0][0]) ? (-100000000) : private_sim_score_ext[0][0]);
      }
      if (t0 >= 1 && t1 >= 1)
        sim_score_ext[t0 * 8 + t1] = private_sim_score_ext[0][0];
    }
}
__global__ void kernel1(int *gap_size_v_ext)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    int private_gap_size_v_ext[1][1];

    {
      private_gap_size_v_ext[0][0] = 1;
      gap_size_v_ext[0 * 8 + t0] = private_gap_size_v_ext[0][0];
    }
}
__global__ void kernel2(int *best_gap_v_ext)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    int private_best_gap_v_ext[1][1];

    {
      private_best_gap_v_ext[0][0] = (-260);
      best_gap_v_ext[0 * 8 + t0] = private_best_gap_v_ext[0][0];
    }
}
__global__ void kernel3(int *gap_size_h_ext)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    int private_gap_size_h_ext[1][1];

    {
      private_gap_size_h_ext[0][0] = 1;
      gap_size_h_ext[t0 * 8 + 0] = private_gap_size_h_ext[0][0];
    }
}
__global__ void kernel4(int *best_gap_h_ext)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    int private_best_gap_h_ext[1][1];

    {
      private_best_gap_h_ext[0][0] = (-260);
      best_gap_h_ext[t0 * 8 + 0] = private_best_gap_h_ext[0][0];
    }
}
__global__ void kernel5(int *H, int *H_ext, int *best_gap_h_ext, int *best_gap_v_ext, int *bt, int *gap_size_h_ext, int *gap_size_v_ext, int *sim_score_ext, int *step_diag_ext, int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    int private_H_ext_6[1][1];
    int private_bt_tmp1_ext[1][1];
    int private_bt_tmp2_ext[1][1];
    int private_kd_ext[1][1];
    int private_ki_ext[1][1];
    int private_step_diag_ext[1][1];
    int private_step_down_ext[1][1];
    int private_step_right_ext[1][1];
    int private_sw_tmp1_ext[1][1];
    int private_sw_tmp2_ext[1][1];

    if (c0 >= t0 && t0 + 7 >= c0) {
      if ((t0 >= 1 && c0 == t0) || t0 == 0) {
        private_step_diag_ext[0][0] = step_diag_ext[t0 * 8 + (-t0 + c0)];
      } else {
        private_step_diag_ext[0][0] = (((-100000000) > (H_ext[(t0 - 1) * 8 + (-t0 + c0 - 1)] + sim_score_ext[t0 * 8 + (-t0 + c0)])) ? (-100000000) : (H_ext[(t0 - 1) * 8 + (-t0 + c0 - 1)] + sim_score_ext[t0 * 8 + (-t0 + c0)]));
      }
      if (c0 >= t0 + 1)
        if ((H_ext[t0 * 8 + (-t0 + c0 - 1)] + (-260)) > (best_gap_h_ext[t0 * 8 + (-t0 + c0 - 1)] + (-11))) {
          best_gap_h_ext[t0 * 8 + (-t0 + c0)] = (H_ext[t0 * 8 + (-t0 + c0 - 1)] + (-260));
          gap_size_h_ext[t0 * 8 + (-t0 + c0)] = 1;
        } else {
          best_gap_h_ext[t0 * 8 + (-t0 + c0)] = (best_gap_h_ext[t0 * 8 + (-t0 + c0 - 1)] + (-11));
          gap_size_h_ext[t0 * 8 + (-t0 + c0)] = (gap_size_h_ext[t0 * 8 + (-t0 + c0 - 1)] + 1);
        }
      if (t0 >= 1)
        if ((H_ext[(t0 - 1) * 8 + (-t0 + c0)] + (-260)) > (best_gap_v_ext[(t0 - 1) * 8 + (-t0 + c0)] + (-11))) {
          best_gap_v_ext[t0 * 8 + (-t0 + c0)] = (H_ext[(t0 - 1) * 8 + (-t0 + c0)] + (-260));
          gap_size_v_ext[t0 * 8 + (-t0 + c0)] = 1;
        } else {
          best_gap_v_ext[t0 * 8 + (-t0 + c0)] = (best_gap_v_ext[(t0 - 1) * 8 + (-t0 + c0)] + (-11));
          gap_size_v_ext[t0 * 8 + (-t0 + c0)] = (gap_size_v_ext[(t0 - 1) * 8 + (-t0 + c0)] + 1);
        }
      private_step_down_ext[0][0] = (((-100000000) > best_gap_v_ext[t0 * 8 + (-t0 + c0)]) ? (-100000000) : best_gap_v_ext[t0 * 8 + (-t0 + c0)]);
      private_kd_ext[0][0] = gap_size_v_ext[t0 * 8 + (-t0 + c0)];
      private_step_right_ext[0][0] = (((-100000000) > best_gap_h_ext[t0 * 8 + (-t0 + c0)]) ? (-100000000) : best_gap_h_ext[t0 * 8 + (-t0 + c0)]);
      private_ki_ext[0][0] = gap_size_h_ext[t0 * 8 + (-t0 + c0)];
      private_sw_tmp1_ext[0][0] = ((private_step_diag_ext[0][0] > private_step_down_ext[0][0]) ? private_step_diag_ext[0][0] : private_step_down_ext[0][0]);
      private_sw_tmp2_ext[0][0] = ((private_sw_tmp1_ext[0][0] > private_step_right_ext[0][0]) ? private_sw_tmp1_ext[0][0] : private_step_right_ext[0][0]);
      private_H_ext_6[0][0] = private_sw_tmp2_ext[0][0];
      H[(t0 + 1) * 9 + (-t0 + c0 + 1)] = private_H_ext_6[0][0];
      private_bt_tmp1_ext[0][0] = ((private_H_ext_6[0][0] == private_step_right_ext[0][0]) ? (-private_ki_ext[0][0]) : private_kd_ext[0][0]);
      private_bt_tmp2_ext[0][0] = ((private_H_ext_6[0][0] == private_step_diag_ext[0][0]) ? 0 : private_bt_tmp1_ext[0][0]);
      bt[t0 * 8 + (-t0 + c0)] = private_bt_tmp2_ext[0][0];
      if (c0 <= 21)
        H_ext[t0 * 8 + (-t0 + c0)] = private_H_ext_6[0][0];
    }
}
